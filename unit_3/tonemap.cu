#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Definition Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/


#include "reference_calc.cpp"
#include "utils.h"
#include <cstdio>
#include <cmath>

#define REDUCE_SHM 1024

template <typename T, typename Op>
__global__ void reduce_inplace(T * const d_buf, unsigned int n, Op const op, T const ne = T())
{
    assert(blockDim.x <= REDUCE_SHM);
    
    unsigned int i = threadIdx.x;
    unsigned int j = threadIdx.x + blockDim.x * blockIdx.x;

    __shared__ T s_buf[REDUCE_SHM];
    
    s_buf[i] = (j < n) ? d_buf[j] : ne;
    
    __syncthreads();
    
    for (unsigned int s = blockDim.x; s > 1; s >>= 1) {
        T x;
        if ((i & 1) == 0)
            x = op(s_buf[i], s_buf[i+1]);
        __syncthreads();
        if ((i & 1) == 0)
            s_buf[i>>1] = x;
        __syncthreads();
    }
    
    if (i == 0) {
        d_buf[blockIdx.x] = s_buf[0];
    }
}

template <typename T, typename Op>
T reduce(T const * const d_input, unsigned int const n, Op const op, T const ne = T())
{
    T * d_buf;
    checkCudaErrors(hipMalloc((void**)&d_buf, sizeof(T)*n));
    checkCudaErrors(hipMemcpy(d_buf, d_input, sizeof(T)*n, hipMemcpyDeviceToDevice));
    
    unsigned int bsz = min(512, n);
    for (unsigned int m = n; m > 1; m = m / bsz + (m%bsz != 0))
        reduce_inplace<T, Op><<<m/bsz + (m%bsz != 0), bsz>>>(d_buf, m, op, ne);
    hipDeviceSynchronize();
    
    T res;
    checkCudaErrors(hipMemcpy(&res, d_buf, sizeof(T), hipMemcpyDeviceToHost));
    hipFree(d_buf);
    return res;
}

template <typename T> struct device_plus { __inline__ __device__ T operator()(T x, T y) const {return x+y;} };
template <typename T> struct device_max { __inline__ __device__ T operator()(T x, T y) const {return max(x,y);} };
template <typename T> struct device_min { __inline__ __device__ T operator()(T x, T y) const {return min(x,y);} };

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */

    /*
    int h_a[10000];
    for (int i = 0; i < 10000; ++i) h_a[i] = i;
    int * d_a;
    hipMalloc((void**)&d_a, 10000*sizeof(int));
    hipMemcpy(d_a, h_a, 10000*sizeof(int), hipMemcpyHostToDevice);
    printf("%d\n", reduce(d_a, 8, device_plus<int>()));
    hipFree(d_a);
    */
    min_logLum = reduce(d_logLuminance, numRows * numCols, device_min<float>(), INFINITY);
    max_logLum = reduce(d_logLuminance, numRows * numCols, device_max<float>(), -INFINITY);
    float range = max_logLum - min_logLum;
}
